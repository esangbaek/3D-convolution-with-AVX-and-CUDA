/***********************************************
 * Intensive programming-2 project-2
 * 3D Convolution
 * GPU (CUDA)
 * Contributor : Yongha Kwon
 ***********************************************/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#define MAX_KERNEL_SIZE 7

 

__constant__ float Mc[MAX_KERNEL_SIZE * MAX_KERNEL_SIZE * MAX_KERNEL_SIZE];

// 2021 11 17 add gpu convolution
__global__ void Conv3D(float *input, float *output, int width, int height, int channel, int kernel_size, int block_size, int tile_size){


    //Ns[block_size][block_size][block_size]
    extern __shared__ float Ns[];
    //printf("shared\n");

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int row_o = blockIdx.y * tile_size + ty;
    int col_o = blockIdx.x * tile_size + tx;
    int ch_o = blockIdx.z * tile_size + tz;

    int row_i = row_o - kernel_size/2;
    int col_i = col_o - kernel_size/2;
    int ch_i = ch_o - kernel_size/2;

    float out = 0.0f;
    if((ch_i < 0) || (ch_i >= channel) || (row_i < 0) || (row_i >= height) || (col_i < 0) || (col_i >= width)){
        //printf("%d\n",tz*block_size*block_size + ty*block_size + tx);
        Ns[tz*block_size*block_size + ty*block_size + tx] = 0.0f;
    }
    else{
        //printf("%d\n",tz*block_size*block_size + ty*block_size + tx);
        Ns[tz*block_size*block_size + ty*block_size + tx] = input[ch_i * height * width + row_i * width + col_i];
        
    }
    __syncthreads();
    //printf("sync end\n");
    
    if(ty < tile_size && tx < tile_size && tz < tile_size){
        for(int i = 0; i < kernel_size; i++){
            for(int j = 0; j < kernel_size; j++){
                for(int k = 0; k < kernel_size; k++){
                    
                    out += Mc[i*kernel_size*kernel_size+j*kernel_size+k] * Ns[(i+tz)*block_size*block_size + (j+ty) * block_size + (k+tx)];
                    //printf("%d %d %d %d %d %d\n", i, j, k, tz, ty, tx);
                }
            }
        }
        // printf("%d %f\n",row_o * width + col_o ,output);
        if(row_o < height && col_o < width && ch_o < channel)
            output[ch_o * height*width + row_o * width + col_o] = out;
    }
}

//2021 11 17 add verification function
void verification(const float *ans, const float *ret, int channel, int height, int width){
    for(int i = 0; i < channel * height * width; i++){
        if(abs(ret[i] - ans[i]) >= 0.001f){
            printf("Not Equal\n");
            return;
        }
    }

    printf("equal\n");
    return;

}


//2021 11 18 add cuda run function
void run_cuda(const float *input_tmp, const float *kernel_tmp, float *ret, const int channel, const int height, const int width, const int kernel_size, const int tile_size){
    float *input;
    float *output;
    int block_size;
    hipEvent_t start, end;
    float time_ms;

    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipMalloc((void**)&input, sizeof(float) * channel * height * width);
    hipMalloc((void**)&output, sizeof(float) * channel * height * width);
    //ret =(float*)malloc(sizeof(float)*channel*height*width);

    hipMemcpyToSymbol(HIP_SYMBOL(Mc), kernel_tmp, sizeof(float) * kernel_size * kernel_size * kernel_size);
    hipMemcpy(input, input_tmp, sizeof(float) * channel * height * width, hipMemcpyHostToDevice);

    block_size = tile_size + (kernel_size - 1);
    //printf("%d\n", block_size);
    printf("\n\ngpu block size: %d, total_size: %d\n",block_size, block_size * block_size*block_size);
    dim3 dimBlock(block_size, block_size, block_size);
    dim3 dimGrid(ceil(width / (tile_size * 1.0)), ceil(height / (tile_size * 1.0)), ceil(channel / (tile_size * 1.0)));

    hipEventRecord(start, 0);
    Conv3D <<< dimGrid, dimBlock, sizeof(float)*block_size*block_size*block_size >>> (input, output, width, height, channel, kernel_size, block_size, tile_size);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time_ms, start, end);

    hipDeviceSynchronize();
    
    hipMemcpy(ret, output, sizeof(float) * channel * height * width, hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		// somthing's gone wrong
		// print out the CUDA error as a string
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));

		// we can't recover from the error -- exit the program
		return;
	}
    printf("time : %f ms\n", time_ms);
}


//2021 11 17 setting and run conv3d
//2021 11 18 modify to not call direct conv3d. instead call run_cuda
int main(int argc, char *argv[]){
    //blcok size 가 11이 되면 되지 않음.

    int channel, width, height, kernel_size, tile_size;
    int i = 0;
    float tmp_data;

    char input_path[30];
    char output_path[30];
    char kernel_path[30];

    strcpy(input_path, argv[1]);
    strcpy(kernel_path, argv[1]);
    strcpy(output_path, argv[1]);

    
    float *input_tmp, *kernel_tmp, *ret;
    float *ans;
    
    tile_size = atoi(argv[2]);
    
    strcat(input_path, "input.txt");
    strcat(output_path, "output.txt");
    strcat(kernel_path, "kernel.txt");

    FILE *fp1 = fopen(input_path,"r");
    if(fp1){
        printf("open file: ");
    }
    else{
        printf("can't open file\n");
        return -1;
    }
    printf("%s\n",input_path);
    fscanf(fp1, "%d %d %d ", &channel, &height, &width);
    input_tmp = (float*)malloc(sizeof(float)*width*height*channel);
    
    ans = (float*)malloc(sizeof(float)*width*height*channel);

    i = 0;
    
    while(fscanf(fp1, "%f ", &tmp_data) > 0){
        input_tmp[i++] = tmp_data;
        //printf("%f\n",tmp_data);
    }
    
    fclose(fp1);

    FILE *fp2 = fopen(output_path, "r");

    fscanf(fp2, "%d %d %d ", &channel, &height, &width);

    i = 0;
    while(fscanf(fp2, "%f ", &tmp_data) > 0){
        ans[i++] = tmp_data;
    }
    fclose(fp2);

    FILE *fp3 = fopen(kernel_path, "r");
    fscanf(fp3, "%d ", &kernel_size);
    kernel_tmp = (float*)malloc(sizeof(float) * kernel_size * kernel_size * kernel_size);
    i = 0;
    while(fscanf(fp3, "%f ", &tmp_data) > 0){
        kernel_tmp[i++] = tmp_data;
    }
    fclose(fp3);

    ret =(float*)malloc(sizeof(float)*channel*height*width);

    if((tile_size + kernel_size - 1) > 10){
        printf("block size is %d((%d)tile_size + ((%d)kernel_size - 1)\n",(tile_size + kernel_size - 1), tile_size, kernel_size);
        printf("block size must less than 11\n");
        return -1;
    }

    run_cuda(input_tmp, kernel_tmp, ret, channel, height, width, kernel_size, tile_size);
    verification(ans, ret, channel, height, width);

    return 0;
}